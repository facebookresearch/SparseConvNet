#include "hip/hip_runtime.h"
// Copyright 2016-present, Facebook, Inc.
// All rights reserved.
//
// This source code is licensed under the BSD-style license found in the
// LICENSE file in the root directory of this source tree.

#include "RuleBookIterator.h"

// NTX must be >=2 so r is filled properly
template <typename T, Int NTX, Int NTY>
__global__ void MaxPooling_fp(T *input_features, T *output_features,
                              Int nPlanes, Int input_stride, Int output_stride,
                              Int *rules, Int nHot) {
  __shared__ Int r[NTY * 2];
  for (Int n = blockIdx.x * NTY; n < nHot; n += gridDim.x * NTY) {
    {
      Int i = threadIdx.x + NTX * threadIdx.y;
      if (i < NTY * 2 and i < 2 * (nHot - n))
        r[i] = rules[2 * n + i];
    }
    __syncthreads();
    if (n + threadIdx.y < nHot) {
      Int i = r[2 * threadIdx.y] * input_stride;
      Int o = r[2 * threadIdx.y + 1] * output_stride;
      for (Int plane = threadIdx.x; plane < nPlanes; plane += NTX) {
        T inp = input_features[i + plane];
        if (output_features[o + plane] < inp)
          output_features[o + plane] = inp;
      }
    }
    __syncthreads();
  }
}

template <typename T>
void cuda_MaxPooling_ForwardPass(T *input_features, T *output_features,
                                 Int nPlanes, Int input_stride,
                                 Int output_stride, RuleBook _rules) {

  auto application = [&](Int *rbB, Int nHotB, hipStream_t &stream) -> void {
    MaxPooling_fp<T, 32, 32><<<32, dim3(32, 32), 0, stream>>>(
        input_features, output_features, nPlanes, input_stride, output_stride,
        rbB, nHotB);
  };

  iterateRuleBookSeq(_rules, application);
}
template <typename T, Int NTX, Int NTY>
__global__ void MaxPooling_bp(T *input_features, T *d_input_features,
                              T *output_features, T *d_output_features,
                              Int nPlanes, Int input_stride, Int output_stride,
                              Int *rules, Int nHot) {
  __shared__ Int r[NTY * 2];
  for (Int n = blockIdx.x * NTY; n < nHot; n += gridDim.x * NTY) {
    {
      Int i = threadIdx.x + NTX * threadIdx.y;
      if (i < NTY * 2 and i < 2 * (nHot - n))
        r[i] = rules[2 * n + i];
    }
    __syncthreads();
    if (n + threadIdx.y < nHot) {
      Int i = r[2 * threadIdx.y] * input_stride;
      Int o = r[2 * threadIdx.y + 1] * output_stride;
      for (Int plane = threadIdx.x; plane < nPlanes; plane += NTX)
        if (output_features[o + plane] == input_features[i + plane])
          d_input_features[i + plane] += d_output_features[o + plane];
    }
    __syncthreads();
  }
}

template <typename T>
void cuda_MaxPooling_BackwardPass(T *input_features, T *d_input_features,
                                  T *output_features, T *d_output_features,
                                  Int nPlanes, Int input_stride,
                                  Int output_stride, RuleBook _rules) {

  auto application = [&](Int *rbB, Int nHotB, hipStream_t &stream) -> void {
    MaxPooling_bp<T, 32, 32><<<32, dim3(32, 32), 0, stream>>>(
      input_features, d_input_features, output_features, d_output_features,
      nPlanes, input_stride, output_stride, rbB, nHotB);
  };

  iterateRuleBookSeq(_rules, application);

}
