// Copyright 2016-present, Facebook, Inc.
// All rights reserved.
//
// This source code is licensed under the BSD-style license found in the
// LICENSE file in the root directory of this source tree.

#include <ATen/ATen.h>
#include <Metadata/Metadata.h>

#include "CUDA/ActivePooling.cu"
#include "CUDA/AffineReluTrivialConvolution.cu"
#include "CUDA/AveragePooling.cu"
#include "CUDA/BatchNormalization.cu"
#include "CUDA/BatchwiseMultiplicativeDropout.cu"
#include "CUDA/Convolution.cu"
#include "CUDA/Deconvolution.cu"
#include "CUDA/IOLayers.cu"
#include "CUDA/LeakyReLU.cu"
#include "CUDA/MaxPooling.cu"
#include "CUDA/SparseToDense.cu"
#include "CUDA/UnPooling.cu"

template void ActivePooling_ForwardPass<float>(float *input_features,
					       float *output_features,
					       Int batchSize, Int maxActive,
					       Int nPlanes, Int *rules,
					       bool average);
template void ActivePooling_BackwardPass<float>(float *d_input_features,
						float *d_output_features,
						Int batchSize, Int maxActive,
						Int nPlanes, Int *rules,
						bool average);

template void dAffineReluTrivialConvolution_forward<float>(
    float *inFeatures, float *outFeatures, float *affineWeight,
    float *affineBias, float *convWeight, Int input_nPlanes, Int input_stride,
    Int output_nPlanes, Int output_stride, Int nActive);
template void dAffineReluTrivialConvolution_backward_dW<float>(
    float *inFeatures, float *dInFeatures, float *dOutFeatures,
    float *affineWeight, float *dAffineWeight, float *affineBias,
    float *dAffineBias, float *convWeight, float *dConvWeight,
    Int input_nPlanes, Int input_stride, Int output_nPlanes, Int output_stride,
    Int nActive, bool additiveGrad);

template void cuda_AveragePooling_ForwardPass<float>(
    float *input_features, float *output_features, Int nPlanes,
    Int input_stride, Int output_stride, RuleBook _rules, Int filterVolume);
template void cuda_AveragePooling_BackwardPass<float>(
    float *d_input_features, float *d_output_features, Int nPlanes,
    Int input_stride, Int output_stride, RuleBook _rules, Int filterVolume);

template void Convolution_fp_bias<float>(float *oF, float *b, Int nPlanes,
					 Int nActive);
template void Convolution_bp_bias<float>(float *d_oF, float *d_b,
					 Int nPlanes, Int nActive);
template double dConvolution_forward2<float>(
    float *inFeatures, float *outFeatures, float *w, RuleBook _rules,
    Int input_nPlanes, Int input_stride, Int output_nPlanes, Int output_stride);

template void dConvolution_backward_dW2<float>(
    float *inFeatures, float *dInFeatures, float *dOutFeatures, float *w,
    float *dw, RuleBook _rules, Int input_nPlanes, Int input_stride,
    Int output_nPlanes, Int output_stride);

template double dDeconvolution_forward2<float>(
    float *inFeatures, float *outFeatures, float *w, RuleBook _rules,
    Int input_nPlanes, Int input_stride, Int output_nPlanes, Int output_stride);

template void dDeconvolution_backward_dW2<float>(
    float *inFeatures, float *dInFeatures, float *dOutFeatures, float *w,
    float *dw, RuleBook _rules, Int input_nPlanes, Int input_stride,
    Int output_nPlanes, Int output_stride);

template void InputLayer_fp<float>(float *input_features,
				   float *output_features, Int nRows,
				   Int maxActive, Int nPlanes, Int *rules_cpu,
				   Int *rules_gpu, bool average);
template void InputLayer_bp<float>(float *d_input_features,
				   float *d_output_features, Int nRows,
				   Int maxActive, Int nPlanes, Int *rules_cpu,
				   Int *rules_gpu, bool average);

template void LeakyReLU_fp<float>(float *input_features, float *output_features,
				  Int n, float alpha);
template void LeakyReLU_bp<float>(float *input_features,
				  float *d_input_features,
				  float *output_features, Int n, float alpha);
template void cuda_MaxPooling_ForwardPass<float>(float *input_features,
						 float *output_features,
						 Int nPlanes, Int input_stride,
						 Int output_stride,
						 RuleBook _rules);
template void cuda_MaxPooling_BackwardPass<float>(
    float *input_features, float *d_input_features, float *output_features,
    float *d_output_features, Int nPlanes, Int input_stride, Int output_stride,
    RuleBook _rules);
template void cuda_SparseToDense_ForwardPass<float>(float *input_features,
						    float *output_features,
						    Int nPlanes,
						    Int spatialVolume,
						    RuleBook _rules);
template void cuda_SparseToDense_BackwardPass<float>(float *d_input_features,
						     float *d_output_features,
						     Int nPlanes,
						     Int spatialVolume,
						     RuleBook _rules);
template void cuda_UnPooling_ForwardPass<float>(float *input_features,
						float *output_features,
						Int nPlanes, Int input_stride,
						Int output_stride,
						RuleBook _rules);
template void cuda_UnPooling_BackwardPass<float>(float *d_input_features,
						 float *d_output_features,
						 Int nPlanes, Int input_stride,
						 Int output_stride,
						 RuleBook _rules);

template void bn_f<float>(float *iF, float *oF, Int nPlanes, Int input_stride,
			  Int output_stride, Int nActive, float *saveMean,
			  float *saveInvStd, float *runningMean,
			  float *runningVar, float *weight, float *bias,
			  float eps, float momentum, bool train,
			  float leakiness);
template void bn_b<float>(float *input_features, float *d_input_features,
			  float *output_features, float *d_output_features,
			  Int nPlanes, Int input_stride, Int output_stride,
			  Int nActive, float *saveMean, float *saveInvStd,
			  float *runningMean, float *runningVar, float *weight,
			  float *bias, float *d_weight, float *d_bias,
			  float leakiness);

template void bmd_f<float>(float *input_features, float *output_features,
			   float *noise, Int nActive, Int nPlanes, float alpha);
template void bmd_b<float>(float *input_features, float *d_input_features,
			   float *d_output_features, float *noise, Int nActive,
			   Int nPlanes, float alpha);

template void cuda_CopyFeaturesHelper_ForwardPass<float>(
	 float* context, float* Context,Int* rules, Int nPlanes, Int nHot);
template void cuda_CopyFeaturesHelper_BackwardPass<float>(
	 float* dcontext, float* dContext,Int* rules, Int nPlanes, Int nHot);